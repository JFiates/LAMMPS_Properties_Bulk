/********************************************************************************/
/* School of Chemical Engineering, University of Campinas, Brazil               */
/* Code to calculate viscosity using Green-Kubo equation   			*/
/* Developer: Dr. Juliane Fiates						*/
/* Paralelization: Leandro Negrini Zanotto					*/
/********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <new>
#include <iostream>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void integrate_kernel(const float *savg, const int *time,
	float *vis_vec, const int n_max, const float cons) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int step;
	float vis = savg[0];
	if (id < n_max) {		
		for (step = 1; step < id - 1; step++)
			vis += 2.0 * savg[step];			
		
		vis_vec[id] = cons * (float) (time[step] - time[0])
				* (vis + savg[step]) / (float) (id - 1);
	}
}

__global__ void stress_acf_kernel(const float *presxy, const float *presxz,
		const float *presyz, const float *presx1, const float *presy1,
		const float *presz1, const int half_n_steps, float *savg) {

	// Get our global thread ID
	int id = blockIdx.x * blockDim.x + threadIdx.x;

    float sxy_n, sxz_n, syz_n, sx1_n, sy1_n, sz1_n;
    float sxy = 0.0, sxz = 0.0, syz = 0.0, sx1 = 0.0, sy1 = 0.0, sz1 = 0.0;
    float f_half_n_steps = float(half_n_steps);
	// Make sure we do not go out of bounds
	if (id < half_n_steps) {
		for (int step = 0; step < half_n_steps; step++) {
			sxy += presxy[step] * presxy[step + id];
			sxz += presxz[step] * presxz[step + id];
			syz += presyz[step] * presyz[step + id];
			sx1 += presx1[step] * presx1[step + id];
			sy1 += presy1[step] * presy1[step + id];
			sz1 += presz1[step] * presz1[step + id];

		}
		sxy_n = sxy / f_half_n_steps;
		sxz_n = sxz / f_half_n_steps;
		syz_n = syz / f_half_n_steps;
		sx1_n = sx1 / f_half_n_steps;
		sy1_n = sy1 / f_half_n_steps;
		sz1_n = sz1 / f_half_n_steps;
		savg[id] =  (sxy_n + sxz_n + syz_n + sx1_n + sy1_n + sz1_n) / 6.0;
	}
	//__syncthreads();

   //integrate_kernel(savg, time, vis_vec, n_max, cons);

}


int main(int argc, char **argv) {
	/************************************************************************/
	/* Variables declaration                                                */
	/************************************************************************/
	/* Counter 								*/
	//int k;
	/* Counter for data points 						*/
	int step;
	/* Total number of steps                                                */
	int n_steps;
	/* Maximum number of steps for integration				*/
	int n_max;
	/* Counter for pressure tensor components 				*/
	int p;
	/* Boltzmann constant in m2.kg/(s2.K) 					*/
	float kb;
	/* Box volume in m3 							*/
	float volume, density;
	/* Absolute temperature in K 						*/
	float t;
	/* CONST = V/kB/T 							*/
	/* Viscosity in cP = mPa.s 						*/
	/* Sum of each pressure tensor component 				*/
	float sum[6] = { };
	/* Average of each pressure tensor component 				*/
	float avg[6] = { };
	/* Auxiliary string for comments in the input file                      */
	char com[1000];
	/* Input file with the tensor components                                */
	char p_file[400];
	/* Output file with the stress auto-correlation function                */
	char out_sacf_file[500];
	/* Output file with the viscosity coefficient as function of time       */
	char vis_file[500];

	double start = omp_get_wtime();

	FILE *in, *out;

	/************************************************************************/
	/* Physical constants                                                   */
	/************************************************************************/
	kb = 1.38064852e-23;

	/************************************************************************/
	/* Reading input file                                                   */
	/************************************************************************/
	in = fopen(argv[1], "r");
	if ((in = fopen(argv[1], "r")) == NULL) {
		cout << "No such file" << "\n" << argv[1];
		exit(1);
	}

	fscanf(in, "%s", com);
	fscanf(in, "%f", &t);
	fscanf(in, "%s", com);
	fscanf(in, "%d", &n_steps);
	fscanf(in, "%s", com);
	fscanf(in, "%d", &n_max);
	fscanf(in, "%s", com);
	fscanf(in, "%s", p_file);
	fscanf(in, "%s", com);
	fscanf(in, "%s", out_sacf_file);
	fscanf(in, "%s", com);
	fscanf(in, "%s", vis_file);
	fclose(in);
	int half_n_steps = n_steps / 2;

	if (n_max > half_n_steps) {
		printf(" Error! n_max must be lower than n_steps/2!");
		return 0;
	}

	/************************************************************************/
	/* Memory allocation                                                    */
	/************************************************************************/
	/* Time in fs 								*/
	int *time = new int[n_steps];

	/* Pressure tensor components on CPU 	    		                */
	float presxx, presyy, preszz;
	float *presxy = new float[n_steps]();
	float *presxz = new float[n_steps]();
	float *presyz = new float[n_steps]();
	float *presx1 = new float[n_steps]();
	float *presy1 = new float[n_steps]();
	float *presz1 = new float[n_steps]();
	/* Average of the stress correlation function 				*/
	float *savg = new float[half_n_steps];
	float *vis_vec = new float[n_max];

	// Size, in bytes, of each vector
	size_t bytes = n_steps * sizeof(float);
	/* Pressure tensor components on GPU 	    		                */
	float *presxy_d = NULL;
	float *presxz_d = NULL;
	float *presyz_d = NULL;
	float *presx1_d = NULL;
	float *presy1_d = NULL;
	float *presz1_d = NULL;
	float *vis_vec_d = NULL;
	int *time_d = NULL;
	/* Average of the stress correlation function 				*/
	float *savg_d;

	/************************************************************************/
	/* Reading pressure tensor components file 				                */
	/************************************************************************/
	in = fopen(p_file, "r");
	cout << "\nReading input file...\n";
	for (int step = 0; step < n_steps; step++) {
		fscanf(in, "%d %f %f %f %f %f %f %f %f", &time[step], &presxx,
				&presxy[step], &presxz[step], &presyy, &presyz[step], &preszz,
				&density, &volume);
		presx1[step] = 0.5 * (presxx - presyy);
		presy1[step] = 0.5 * (presyy - preszz);
		presz1[step] = 0.5 * (presxx - preszz);
	}
	fclose(in);
	volume = volume * 1e-30;
	float cons = (0.010266755 * volume / kb / t) * 0.5;
	
	/************************************************************************/
	/* Normalization of the pressure tensor components 	 		*/
	/************************************************************************/
	for (int step = 0; step < n_steps; step++) {
		sum[0] += presx1[step];
		sum[1] += presxy[step];
		sum[2] += presxz[step];
		sum[3] += presy1[step];
		sum[4] += presyz[step];
		sum[5] += presz1[step];
	}

	for (p = 0; p < 6; p++)
		avg[p] = sum[p] / (float) (n_steps);

	for (int k = 0; k < n_steps; k++) {
		presx1[k] -= avg[0];
		presxy[k] -= avg[1];
		presxz[k] -= avg[2];
		presy1[k] -= avg[3];
		presyz[k] -= avg[4];
		presz1[k] -= avg[5];
	}

	int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
	    printf("hipGetDeviceCount returned %d\n-> %s\n",
            static_cast<int>(error_id), hipGetErrorString(error_id));
	    printf("Result = FAIL\n");
	    exit(EXIT_FAILURE);
	}

 	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
	    printf("There are no available device(s) that support CUDA the Code will run on CPU\n");
	} else {
	    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
        }
	
	if (deviceCount != 0) {
		

		// Allocate memory for each vector on GPU
		hipMalloc(&presxy_d, bytes);
		hipMalloc(&presxz_d, bytes);
		hipMalloc(&presyz_d, bytes);
		hipMalloc(&presx1_d, bytes);
		hipMalloc(&presy1_d, bytes);
		hipMalloc(&presz1_d, bytes);
		hipMalloc(&vis_vec_d, n_max * sizeof(float));
		hipMalloc(&time_d, n_steps * sizeof(int));
		hipMalloc(&savg_d, half_n_steps * sizeof(float));


		/************************************************************************/
		/* Copying Arrays from host to device			 		*/
		/************************************************************************/
		hipMemcpy(presxy_d, presxy, bytes, hipMemcpyHostToDevice);
		hipMemcpy(presxz_d, presxz, bytes, hipMemcpyHostToDevice);
		hipMemcpy(presyz_d, presyz, bytes, hipMemcpyHostToDevice);
		hipMemcpy(presx1_d, presx1, bytes, hipMemcpyHostToDevice);
		hipMemcpy(presy1_d, presy1, bytes, hipMemcpyHostToDevice);
		hipMemcpy(presz1_d, presz1, bytes, hipMemcpyHostToDevice);
		hipMemcpy(time_d, time, n_steps * sizeof(int), hipMemcpyHostToDevice);

		/************************************************************************/
		/* Stress Auto-Correlation Function calculation                         */
		/************************************************************************/

		cout << "Calculating Stress ACF on GPU and Integrating on GPU...\n";

		int threadsPerBlock = 1024;
		int blocksPerGrid = (half_n_steps + threadsPerBlock - 1) / threadsPerBlock;

		// Execute the kernel
		stress_acf_kernel<<<blocksPerGrid, threadsPerBlock>>>(presxy_d, presxz_d, presyz_d,
				presx1_d, presy1_d, presz1_d, half_n_steps, savg_d);

		blocksPerGrid = (n_max + threadsPerBlock - 1) / threadsPerBlock;
		integrate_kernel<<<blocksPerGrid, threadsPerBlock>>>(savg_d, time_d, vis_vec_d, n_max, cons);


		hipDeviceSynchronize();
		/** Copying the array back to write the file **/
		hipMemcpy(savg, savg_d, half_n_steps * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(vis_vec, vis_vec_d, n_max * sizeof(float), hipMemcpyDeviceToHost);
	} else {


	//Run on CPU in Parallel
	//Stress correlation functions
	int numthreads = omp_get_num_threads();
	cout << "Calculating Stress ACF on CPU using " << "\t" << numthreads << "\n";
	float f_half_n_steps = (float) half_n_steps;
	float sxy_n = 0.0, sxz_n = 0.0, syz_n = 0.0, sx1_n = 0.0, sy1_n = 0.0, sz1_n = 0.0;
		 #pragma omp parallel for private(sxy_n,sxz_n,syz_n,sx1_n,sy1_n,sz1_n)
		 for (int k = 0; k < half_n_steps; k++){
			 float sxy = 0.0, sxz = 0.0, syz = 0.0, sx1 = 0.0, sy1 = 0.0, sz1 = 0.0;
			 for (int step = 0; step < half_n_steps; step++){
				 sxy += presxy[step] * presxy[step + k];
				 sxz += presxz[step] * presxz[step + k];
				 syz += presyz[step] * presyz[step + k];
				 sx1 += presx1[step] * presx1[step + k];
				 sy1 += presy1[step] * presy1[step + k];
				 sz1 += presz1[step] * presz1[step + k];
			 }
		 sxy_n = sxy / f_half_n_steps;
		 sxz_n = sxz / f_half_n_steps;
		 syz_n = syz / f_half_n_steps;
		 sx1_n = sx1 / f_half_n_steps;
		 sy1_n = sy1 / f_half_n_steps;
		 sz1_n = sz1 / f_half_n_steps;
		 savg[k] = (sxy_n + sxz_n + syz_n + sx1_n + sy1_n + sz1_n)/6.0;
	 }


	/************************************************************************/
	/* Integration of Stress Auto-Correlation Function 		 	*/
	/* Green-Kubo equation for viscosity                                    */
	/************************************************************************/
	cout << "Integrating Stress ACF to calculate viscosity on CPU...\n";

	 // Run on CPU in Parallel	  
	 #pragma omp parallel for private(step)
	 for (int k = 0; k < n_max; k++){
		 float vis = savg[0];
		 for (step = 1; step < k-1; step++){
			 vis += 2.0 * savg[step];			 
		 }
	 	 vis_vec[k] = cons * (float)(time[step] - time[0]) * (vis + savg[step]) / (float)(k-1);
	 }
	}	

 	cout << "Writing the Files...\n";
	out = fopen(out_sacf_file, "w");
	for (int k = 0; k < half_n_steps; k++)
	    fprintf(out, "%d %f %f\n", time[k]-time[0], savg[k], savg[k]/savg[0]);
	fclose(out);

	 out = fopen(vis_file, "w");
	 for (int k = 3; k < n_max; k++){
		 for (step = 1; step < k-1; step++){}
		 fprintf(out, "%d %f\n", time[step]-time[0], vis_vec[k]);
	 }
	 fclose(out);

	//Free Memory from CPU and GPU
	delete[] presxy;
	delete[] presxz;
	delete[] presyz;
	delete[] presx1;
	delete[] presy1;
	delete[] presz1;
	delete[] savg;
	delete[] vis_vec;

	if (deviceCount != 0) {
		hipFree(presxy_d);
		hipFree(presxz_d);
		hipFree(presyz_d);
		hipFree(presx1_d);
		hipFree(presy1_d);
		hipFree(presz1_d);
		hipFree(savg_d);
		hipFree(vis_vec_d);
		hipFree(time_d);
	}
	double end = omp_get_wtime();
	cout << "The calculation is ended...\n";
	cout << "Elapsed Time: " << (end - start) << "s";

	return 0;

}

